#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <iostream>
#include <sys/time.h> // Not implemented in Windows
#include <complex>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cublasHelper.hpp"
#include "cudaMultiShiftTrsm.hpp"

using namespace std;

// ===============================================
// BLAS and LAPACK routines
// ===============================================
extern "C" 
{
  float snrm2_(int *n, void *x, int *incx);
  double dnrm2_(int *n, void *x, int *incx);
  float scnrm2_(int *n, void *x, int *incx);
  double dznrm2_(int *n, void *x, int *incx);
  void saxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void daxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void caxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void zaxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void ssyrk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void dsyrk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void cherk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void zherk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void strmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void dtrmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void ctrmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void ztrmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void spotrf_(char *uplo, int *n, void *A, int *lda, int *info);
  void dpotrf_(char *uplo, int *n, void *A, int *lda, int *info);
  void cpotrf_(char *uplo, int *n, void *A, int *lda, int *info);
  void zpotrf_(char *uplo, int *n, void *A, int *lda, int *info);
}
template <typename F> inline
double nrm2(int n, F * x, int incx);
template <> inline
double nrm2<float>(int n, float * x, int incx) {
  return snrm2_(&n,x,&incx);
}
template <> inline
double nrm2<double>(int n, double * x, int incx) {
  return dnrm2_(&n,x,&incx);
}
template <> inline
double nrm2<complex<float> >(int n, complex<float> * x, int incx) {
  return scnrm2_(&n,x,&incx);
}
template <> inline
double nrm2<complex<double> >(int n, complex<double> * x, int incx) {
  return dznrm2_(&n,x,&incx);
}
template <typename F> inline
void axpy(int n, F a, F * x, int incx, F * y, int incy);
template <> inline
void axpy<float>(int n, float a, float * x, int incx,
		 float * y, int incy) {
  saxpy_(&n,&a,x,&incx,y,&incy);
}
template <> inline
void axpy<double>(int n, double a, double * x, int incx,
		  double * y, int incy) {
  daxpy_(&n,&a,x,&incx,y,&incy);
}
template <> inline
void axpy<complex<float> >(int n, complex<float> a,
			   complex<float> * x, int incx,
			   complex<float> * y, int incy) {
  caxpy_(&n,&a,x,&incx,y,&incy);
}
template <> inline
void axpy<complex<double> >(int n, complex<double> a,
			    complex<double> * x, int incx,
			    complex<double> * y, int incy) {
  zaxpy_(&n,&a,x,&incx,y,&incy);
}
template <typename F> inline
void herk(char uplo, char trans, int n, int k,
	  F alpha, F * A, int lda,
	  F beta, F * C, int ldc);
template <> inline
void herk<float>(char uplo, char trans, int n, int k,
		 float alpha, float * A, int lda,
		 float beta, float * C, int ldc) {
  ssyrk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <> inline
void herk<double>(char uplo, char trans, int n, int k,
		  double alpha, double * A, int lda,
		  double beta, double * C, int ldc) {
  dsyrk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <> inline
void herk<complex<float> >(char uplo, char trans, int n, int k,
			   complex<float> alpha,
			   complex<float> * A, int lda,
			   complex<float> beta,
			   complex<float> * C, int ldc) {
  cherk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <> inline
void herk<complex<double> >(char uplo, char trans, int n, int k,
			    complex<double> alpha,
			    complex<double> * A, int lda,
			    complex<double> beta,
			    complex<double> * C, int ldc) {
  zherk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <typename F> inline
void trmm(char side, char uplo, char transa, char diag, 
	  int n, int m, F alpha, F * A, int lda,
	  F * B, int ldb);
template <> inline
void trmm<float>(char side, char uplo, char transa, char diag, 
		 int n, int m, float alpha, float * A, int lda,
		 float * B, int ldb) {
  strmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <> inline
void trmm<double>(char side, char uplo, char transa, char diag, 
		  int n, int m, double alpha, double * A, int lda,
		  double * B, int ldb) {
  dtrmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <> inline
void trmm<complex<float> >(char side, char uplo, char transa, char diag, 
			   int n, int m, complex<float> alpha,
			   complex<float> * A, int lda,
			   complex<float> * B, int ldb) {
  ctrmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <> inline
void trmm<complex<double> >(char side, char uplo, char transa, char diag, 
			    int n, int m, complex<double> alpha,
			    complex<double> * A, int lda,
			    complex<double> * B, int ldb) {
  ztrmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <typename F> inline
void potrf(char uplo, int n, F * A, int lda, int &info);
template <> inline
void potrf<float>(char uplo, int n, float * A, int lda, int &info) {
  spotrf_(&uplo,&n,A,&lda,&info);
}
template <> inline
void potrf<double>(char uplo, int n, double * A, int lda, int &info) {
  dpotrf_(&uplo,&n,A,&lda,&info);
}
template <> inline
void potrf<complex<float> >(char uplo, int n,
			    complex<float> * A, int lda, int &info) {
  cpotrf_(&uplo,&n,A,&lda,&info);
}
template <> inline
void potrf<complex<double> >(char uplo, int n,
			     complex<double> * A, int lda, int &info) {
  zpotrf_(&uplo,&n,A,&lda,&info);
}

// ===============================================
// Test matrices
// ===============================================

/// Compute Gaussian random variable
/** Uses Box-Muller transform to convert uniform distribution to
 *  Gaussian distribution
 */
template <typename F>
F randn() {
  F u1 = ((F)std::rand())/RAND_MAX;
  F u2 = ((F)std::rand())/RAND_MAX;
  return std::sqrt(-2*std::log(u1))*std::cos(2*M_PI*u2);
}
template <>
complex<float> randn<complex<float> >() {
  float u1 = ((float)std::rand())/RAND_MAX;
  float u2 = ((float)std::rand())/RAND_MAX;
  return complex<float>(std::sqrt(-2*std::log(u1))*std::cos(2*M_PI*u2),
			std::sqrt(-2*std::log(u1))*std::sin(2*M_PI*u2));
}
template <>
complex<double> randn<complex<double> >() {
  double u1 = ((double)std::rand())/RAND_MAX;
  double u2 = ((double)std::rand())/RAND_MAX;
  return complex<double>(std::sqrt(-2*std::log(u1))*std::cos(2*M_PI*u2),
			 std::sqrt(-2*std::log(u1))*std::sin(2*M_PI*u2));
}

/// Generate matrix with Gaussian random variables
template <typename F>
void randn(int n, F *A) {
#pragma omp parallel for
  for(int i=0;i<n;++i)
    A[i] = randn<F>();
}

/// Generate matrix with Cholesky factorization of random matrix
template <typename F>
void choleskyRandomMatrix(char uplo, int m, F *A) {

  // Generate matrix with Gaussian random variables
  F *temp = (F*) std::malloc(m*m*sizeof(F));
  randn<F>(m*m,temp);

  // Construct positive definite matrix
  herk<F>(uplo,'N',m,m,1,temp,m,0,A,m);

  // Shift diagonal to improve condition number
#pragma omp parallel for
  for(int i=0;i<m;++i)
    A[i+i*m] += std::sqrt(m);

  // Perform Cholesky factorization
  int info;
  potrf<F>(uplo, m, A, m, info);

  // Clean up
  free(temp);

}

/// Output matrix to stream
template <typename F>
void printMatrix(ostream & os,
		 const char uplo, const char diag,
		 const int m, const int n, 
		 const F * A, const int lda) {

  os << "    [[";
  for(int i=0;i<m;++i) {
    if(std::toupper(uplo) == 'L') {
      for(int j=0;j<i;++j)
	os << A[i+j*lda] << " ";
      if(std::toupper(diag)=='U')
	os << "1 ";
      else
	os << A[i+i*lda] << " ";
      for(int j=i+1;j<n;++j)
	os << "0 ";
    }
    else if(std::toupper(uplo) == 'U') {
      for(int j=0;j<i;++j)
	os << "0 ";
      if(std::toupper(diag)=='U')
	os << "1 ";
      else
	os << A[i+i*lda] << " ";
      for(int j=i+1;j<n;++j)
	os << A[i+j*lda] << " ";
    }
    else {
      for(int j=0;j<n;++j)
	os << A[i+j*lda] << " ";
    }
    os << "]";
    if(i<m-1)
      os << "\n" << "    [";
    else
      os << "]\n";
  }
}

// ===============================================
// Validation program
// ===============================================

template <typename F>
void validation(const int m, const int n,
		const char side, const char uplo,
		const char trans, const char diag,
		const bool verbose) {

  // -------------------------------------------------
  // Initialization
  // -------------------------------------------------

  // Initialize timing
  timeval timeStart, timeEnd;

  // Initialize memory on host
  F *A = (F*) std::malloc(m*m*sizeof(F));
  F *B = (F*) std::malloc(m*n*sizeof(F));
  F *shifts = (F*) std::malloc(n*sizeof(F));
  F *X = (F*) std::malloc(m*n*sizeof(F));
  F *residual = (F*) std::malloc(m*n*sizeof(F));

  // Initialize matrices on host
  F alpha = randn<F>();
  choleskyRandomMatrix<F>(uplo,m,A);
  randn<F>(m*n,B);
  randn<F>(n,shifts);

  // Initialize memory on device
  F *cuda_A, *cuda_B, *cuda_B_cublas, *cuda_shifts;
  hipMalloc(&cuda_A, m*m*sizeof(F));
  hipMalloc(&cuda_B, m*n*sizeof(F));
  hipMalloc(&cuda_B_cublas, m*n*sizeof(F));
  hipMalloc(&cuda_shifts, n*sizeof(F));
  hipMemcpy(cuda_A, A, m*m*sizeof(F), hipMemcpyHostToDevice);
  hipMemcpy(cuda_B, B, m*n*sizeof(F), hipMemcpyHostToDevice);
  hipMemcpy(cuda_B_cublas, B, m*n*sizeof(F), hipMemcpyHostToDevice);
  hipMemcpy(cuda_shifts, shifts, n*sizeof(F), hipMemcpyHostToDevice);

  // Initialize cuBLAS
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSideMode_t  cublasSide  = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t  cublasUplo  = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t cublasTrans = HIPBLAS_OP_N;
  hipblasDiagType_t  cublasDiag  = HIPBLAS_DIAG_NON_UNIT;
  if(std::toupper(side)=='R')
    cublasSide = HIPBLAS_SIDE_RIGHT;
  if(std::toupper(uplo)=='U')
    cublasUplo = HIPBLAS_FILL_MODE_UPPER;
  if(std::toupper(trans)=='T')
    cublasTrans = HIPBLAS_OP_T;
  else if(std::toupper(trans)=='C') 
    cublasTrans = HIPBLAS_OP_C;
  if(std::toupper(diag)=='U')
    cublasDiag = HIPBLAS_DIAG_UNIT;

  // -------------------------------------------------
  // Test cudaMultiShiftTrsm
  // -------------------------------------------------

  // Solve shifted triangular system
  hipDeviceSynchronize();
  gettimeofday(&timeStart, NULL);
  status = cudaMstrsm::cudaMultiShiftTrsm<F>
    (handle, cublasSide, cublasUplo, cublasTrans, cublasDiag,
     m, n, &alpha, cuda_A, m, cuda_B, m, cuda_shifts);
  hipDeviceSynchronize();
  gettimeofday(&timeEnd, NULL);
  double cudaMstrsmTime
    = timeEnd.tv_sec - timeStart.tv_sec
    + (timeEnd.tv_usec - timeStart.tv_usec)/1e6;
  if(status != HIPBLAS_STATUS_SUCCESS)
    std::cout << "\n" 
	      << "----------------------------------------\n"
	      << "WARNING: cudaMultiStreamTrsm failed\n"
	      << "----------------------------------------\n";

  // Transfer result to host
  hipMemcpy(X, cuda_B, m*n*sizeof(F), 
	     hipMemcpyDeviceToHost);

  // -------------------------------------------------
  // cuBLAS triangular solve
  //   For performance comparison
  // -------------------------------------------------

  // Solve triangular system
  hipDeviceSynchronize();
  gettimeofday(&timeStart, NULL);
  status = cublasTrsm(handle,cublasSide,cublasUplo,cublasTrans,cublasDiag,
		      m,n,&alpha,cuda_A,m,cuda_B_cublas,m);
  hipDeviceSynchronize();
  gettimeofday(&timeEnd, NULL);
  double cublasTime
    = timeEnd.tv_sec - timeStart.tv_sec
    + (timeEnd.tv_usec - timeStart.tv_usec)/1e6;
  if(status != HIPBLAS_STATUS_SUCCESS)
    std::cout << "\n"
	      << "----------------------------------------\n"
	      << "WARNING: cublasTrsm failed\n"
	      << "----------------------------------------\n";

  // -------------------------------------------------
  // Output results
  // -------------------------------------------------

  // Report time for matrix multiplication
  std::cout << "\n"
	    << "Timings\n"
	    << "----------------------------------------\n"
	    << "  cudaMstrsm : " << cudaMstrsmTime << " sec\n"
	    << "  cuBLAS     : " << cublasTime     << " sec\n";

  // Report FLOPS
  double gflopCount = 1e-9*m*m*n; // Approximate
  std::cout << "\n"
	    << "Performance\n"
	    << "----------------------------------------\n"
	    << "  cudaMstrsm : " << gflopCount/cudaMstrsmTime << " GFLOPS\n"
	    << "  cuBLAS     : " << gflopCount/cublasTime     << " GFLOPS\n";
  
  if(verbose) {
    // Print matrices
    std::cout << "\n"
	      << "Matrix entries\n"
	      << "----------------------------------------\n"
	      << "  alpha = " << alpha << "\n";
    std::cout << "  shifts =\n";
    printMatrix<F>(std::cout,'N','N',1,n,shifts,1);
    std::cout << "  A =\n";
    printMatrix<F>(std::cout,uplo,diag,m,m,A,m);
    std::cout << "  B =\n";
    printMatrix<F>(std::cout,'N','N',m,n,B,m);
    std::cout << "  X =\n";
    printMatrix<F>(std::cout,'N','N',m,n,X,m);
  }

  // Check error in solution
  double normB = nrm2<F>(m*n,B,1);
  std::memcpy(residual,X,m*n*sizeof(F));
  trmm<F>(side,uplo,trans,diag,m,n,1,A,m,residual,m);
#pragma omp parallel for
  for(int i=0;i<n;++i)
    axpy<F>(m, shifts[i], X+i*m, 1, residual+i*m, 1);
  axpy<F>(m*n, -alpha, B, 1, residual, 1);
  double relResidual = nrm2<F>(m*n,residual,1)/normB;
  std::cout << "\n"
	    << "Relative error (Frobenius norm)\n"
	    << "----------------------------------------\n"
	    << "  cudaMstrsm : " << relResidual << "\n";

  // -------------------------------------------------
  // Clean up
  // -------------------------------------------------
  std::free(A);
  std::free(B);
  std::free(shifts);
  std::free(X);
  std::free(residual);
  hipFree(cuda_A);
  hipFree(cuda_B);
  hipFree(cuda_B_cublas);
  hipFree(cuda_shifts);
  hipblasDestroy(handle);

}

/// Main function
int main(int argc, char **argv) {
  
  // Default parameters
  int  m        = 4;
  int  n        = 1;
  char dataType = 'S';
  char side     = 'L';
  char uplo     = 'L';
  char trans    = 'N';
  char diag     = 'N';
  bool verbose  = false;

  // User-provided parameters
  if(argc > 1)
    m = std::atoi(argv[1]);
  if(argc > 2)
    n = std::atoi(argv[2]);
  if(argc > 3)
    dataType = std::toupper(argv[3][0]);
  if(argc > 4)
    side = std::toupper(argv[4][0]);
  if(argc > 5)
    uplo = std::toupper(argv[5][0]);
  if(argc > 6)
    trans = std::toupper(argv[6][0]);
  if(argc > 7)
    diag = std::toupper(argv[7][0]);
  if(argc > 8)
    verbose = std::atoi(argv[8]);

  // Report parameters
  std::cout << "========================================\n"
	    << "  SHIFTED TRIANGULAR SOLVE VALIDATION\n"
	    << "========================================\n"
	    << "m = " << m << "\n"
	    << "n = " << n << "\n"
	    << "\n"
	    << "BLAS Options\n"
	    << "----------------------------------------\n"
	    << "Data type = " << dataType << "\n"
	    << "side      = " << side << "\n"
	    << "uplo      = " << uplo << "\n"
	    << "trans     = " << trans << "\n"
	    << "diag      = " << diag << "\n";

  // Perform validation
  if(dataType == 'S')
    validation<float>(m,n,side,uplo,trans,diag,verbose);
  else if(dataType == 'D')
    validation<double>(m,n,side,uplo,trans,diag,verbose);
  else if(dataType == 'C')
    validation<complex<float> >(m,n,side,uplo,trans,diag,verbose);
  else if(dataType == 'Z')
    validation<complex<double> >(m,n,side,uplo,trans,diag,verbose);
  else
    std::cout << "\n" 
	      << "----------------------------------------\n"
	      << "WARNING: Invalid data type\n"
	      << "----------------------------------------\n";

  // Exit
  return EXIT_SUCCESS;

}
