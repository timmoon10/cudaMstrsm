#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <complex>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cudaMultiShiftTrsm.hpp"

#define datafloat float
#define IDX(i,j,ld) ((i)+(j)*(ld))

using namespace std;

// ===============================================
// BLAS and LAPACK routines
// ===============================================
extern "C" 
{
  float snrm2_(int *n, void *x, int *incx);
  double dnrm2_(int *n, void *x, int *incx);
  float scnrm2_(int *n, void *x, int *incx);
  double dznrm2_(int *n, void *x, int *incx);
  void saxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void daxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void caxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void zaxpy_(int *n, void *a, void *x, int *incx, void *y, int *incy);
  void ssyrk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void dsyrk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void cherk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void zherk_(char *uplo, char *trans, int *n, int *k,
	      void *alpha, void *A, int *lda,
	      void *beta, void *C, int *ldc);
  void strmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void dtrmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void ctrmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void ztrmm_(char *side, char *uplo, char *transa, char *diag, 
	      int *n, int *m, void *alpha, void *A, int *lda,
	      void *B, int *ldb);
  void spotrf_(char *uplo, int *n, void *A, int *lda, int *info);
  void dpotrf_(char *uplo, int *n, void *A, int *lda, int *info);
  void cpotrf_(char *uplo, int *n, void *A, int *lda, int *info);
  void zpotrf_(char *uplo, int *n, void *A, int *lda, int *info);
}
template <typename F> inline
double nrm2(int n, F * x, int incx);
template <> inline
double nrm2<float>(int n, float * x, int incx) {
  return snrm2_(&n,x,&incx);
}
template <> inline
double nrm2<double>(int n, double * x, int incx) {
  return dnrm2_(&n,x,&incx);
}
template <> inline
double nrm2<complex<float> >(int n, complex<float> * x, int incx) {
  return scnrm2_(&n,x,&incx);
}
template <> inline
double nrm2<complex<double> >(int n, complex<double> * x, int incx) {
  return dznrm2_(&n,x,&incx);
}
template <typename F> inline
void axpy(int n, F a, F * x, int incx, F * y, int incy);
template <> inline
void axpy<float>(int n, float a, float * x, int incx, float * y, int incy) {
  saxpy_(&n,&a,x,&incx,y,&incy);
}
template <> inline
void axpy<double>(int n, double a, double * x, int incx, double * y, int incy) {
  daxpy_(&n,&a,x,&incx,y,&incy);
}
template <> inline
void axpy<complex<float> >(int n, complex<float> a, complex<float> * x, int incx,
			   complex<float> * y, int incy) {
  caxpy_(&n,&a,x,&incx,y,&incy);
}
template <> inline
void axpy<complex<double> >(int n, complex<double> a, complex<double> * x, int incx,
			    complex<double> * y, int incy) {
  zaxpy_(&n,&a,x,&incx,y,&incy);
}
template <typename F> inline
void herk(char uplo, char trans, int n, int k,
	  F alpha, F * A, int lda,
	  F beta, F * C, int ldc);
template <> inline
void herk<float>(char uplo, char trans, int n, int k,
		 float alpha, float * A, int lda,
		 float beta, float * C, int ldc) {
  ssyrk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <> inline
void herk<double>(char uplo, char trans, int n, int k,
		  double alpha, double * A, int lda,
		  double beta, double * C, int ldc) {
  dsyrk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <> inline
void herk<complex<float> >(char uplo, char trans, int n, int k,
			   complex<float> alpha,
			   complex<float> * A, int lda,
			   complex<float> beta,
			   complex<float> * C, int ldc) {
  cherk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <> inline
void herk<complex<double> >(char uplo, char trans, int n, int k,
			    complex<double> alpha,
			    complex<double> * A, int lda,
			    complex<double> beta,
			    complex<double> * C, int ldc) {
  cherk_(&uplo,&trans,&n,&k,&alpha,A,&lda,&beta,C,&ldc);
}
template <typename F> inline
void trmm(char side, char uplo, char transa, char diag, 
	  int n, int m, F alpha, F * A, int lda,
	  F * B, int ldb);
template <> inline
void trmm<float>(char side, char uplo, char transa, char diag, 
		 int n, int m, float alpha, float * A, int lda,
		 float * B, int ldb) {
  strmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <> inline
void trmm<double>(char side, char uplo, char transa, char diag, 
		  int n, int m, double alpha, double * A, int lda,
		  double * B, int ldb) {
  dtrmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <> inline
void trmm<complex<float> >(char side, char uplo, char transa, char diag, 
			   int n, int m, complex<float> alpha, complex<float> * A, int lda,
			   complex<float> * B, int ldb) {
  ctrmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <> inline
void trmm<complex<double> >(char side, char uplo, char transa, char diag, 
			    int n, int m, complex<double> alpha, complex<double> * A, int lda,
			    complex<double> * B, int ldb) {
  ztrmm_(&side, &uplo, &transa, &diag, &n, &m, &alpha, A, &lda, B, &ldb);
}
template <typename F> inline
void potrf(char uplo, int n, F * A, int lda, int &info);
template <> inline
void potrf<float>(char uplo, int n, float * A, int lda, int &info) {
  spotrf_(&uplo,&n,A,&lda,&info);
}
template <> inline
void potrf<double>(char uplo, int n, double * A, int lda, int &info) {
  dpotrf_(&uplo,&n,A,&lda,&info);
}
template <> inline
void potrf<complex<float> >(char uplo, int n, complex<float> * A, int lda, int &info) {
  cpotrf_(&uplo,&n,A,&lda,&info);
}
template <> inline
void potrf<complex<double> >(char uplo, int n, complex<double> * A, int lda, int &info) {
  zpotrf_(&uplo,&n,A,&lda,&info);
}
template <typename F> inline
hipblasStatus_t cublasTrsm(hipblasHandle_t handle,
			  hipblasSideMode_t side, hipblasFillMode_t uplo,
			  hipblasOperation_t trans, hipblasDiagType_t diag,
			  int m, int n,
			  const F *alpha,
			  const F *A, int lda,
			  F * B, int ldb);
template <> inline
hipblasStatus_t cublasTrsm<float>(hipblasHandle_t handle,
				 hipblasSideMode_t side, hipblasFillMode_t uplo,
				 hipblasOperation_t trans, hipblasDiagType_t diag,
				 int m, int n,
				 const float *alpha,
				 const float *A, int lda,
				 float * B, int ldb) {
  return hipblasStrsm(handle,side,uplo,trans,diag,m,n,alpha,A,lda,B,ldb);
}
template <> inline
hipblasStatus_t cublasTrsm<double>(hipblasHandle_t handle,
				  hipblasSideMode_t side, hipblasFillMode_t uplo,
				  hipblasOperation_t trans, hipblasDiagType_t diag,
				  int m, int n,
				  const double *alpha,
				  const double *A, int lda,
				  double * B, int ldb) {
  return hipblasDtrsm(handle,side,uplo,trans,diag,m,n,alpha,A,lda,B,ldb);
}
template <> inline
hipblasStatus_t cublasTrsm<complex<float> >(hipblasHandle_t handle,
					   hipblasSideMode_t side, hipblasFillMode_t uplo,
					   hipblasOperation_t trans, hipblasDiagType_t diag,
					   int m, int n,
					   const complex<float> *alpha,
					   const complex<float> *A, int lda,
					   complex<float> * B, int ldb) {
  return hipblasCtrsm(handle,side,uplo,trans,diag,m,n,
		     (hipComplex*)alpha,(hipComplex*)A,lda,(hipComplex*)B,ldb);
}
template <> inline
hipblasStatus_t cublasTrsm<complex<double> >(hipblasHandle_t handle,
					    hipblasSideMode_t side, hipblasFillMode_t uplo,
					    hipblasOperation_t trans, hipblasDiagType_t diag,
					    int m, int n,
					    const complex<double> *alpha,
					    const complex<double> *A, int lda,
					    complex<double> * B, int ldb) {
  return hipblasZtrsm(handle,side,uplo,trans,diag,m,n,
		     (hipDoubleComplex*)alpha,(hipDoubleComplex*)A,lda,(hipDoubleComplex*)B,ldb);
}

// ===============================================
// Random matrix generation
// ===============================================

/// Compute Gaussian random variable
/** Uses Box-Muller transform to convert uniform distribution to
 *  Gaussian distribution
 */
template <typename F>
F randn() {
  F u1 = ((F)rand())/RAND_MAX;
  F u2 = ((F)rand())/RAND_MAX;
  return sqrt(-2*log(u1))*cos(2*M_PI*u2);
}
template <>
complex<float> randn<complex<float> >() {
  float u1 = ((float)rand())/RAND_MAX;
  float u2 = ((float)rand())/RAND_MAX;
  return complex<float>(sqrt(-2*log(u1))*cos(2*M_PI*u2),
			sqrt(-2*log(u1))*sin(2*M_PI*u2));
}
template <>
complex<double> randn<complex<double> >() {
  double u1 = ((double)rand())/RAND_MAX;
  double u2 = ((double)rand())/RAND_MAX;
  return complex<double>(sqrt(-2*log(u1))*cos(2*M_PI*u2),
			 sqrt(-2*log(u1))*sin(2*M_PI*u2));
}

/// Generate matrix with Gaussian random variables
/** Diagonal entries are increased to improve conditioning. Viswanath
 *  and Trefethen (1998) find that the lower triangle of this matrix
 *  has a condition number on the order of 2^m.
 */
template <typename F>
void gaussianRandomMatrix(int m, F *A) {
#pragma omp parallel for
  for(int i=0;i<m*m;++i)
    A[i] = randn<F>();
}

/// Generate matrix with Cholesky factorization of random matrix
template <typename F>
void choleskyRandomMatrix(int m, F *A) {

  F *B = (F*) malloc(m*m*sizeof(F));

  // Generate matrix with Gaussian random variables
  gaussianRandomMatrix<F>(m,B);

  // Construct positive definite matrix
  herk<F>('L','N',m,m,1,B,m,0,A,m);

  // Perform Cholesky factorization
  int info;
  potrf<F>('L', m, A, m, info);

  // Clean up
  free(B);

}

// ===============================================
// Validation program
// ===============================================

/// Main function
int main(int argc, char **argv) {
  
  // -------------------------------------------------
  // Initialization
  // -------------------------------------------------

  // Default parameters
  int  m           = 4;
  int  n           = 1;
  bool sideLeft    = true;
  bool lower       = true;
  bool noTranspose = true;
  bool nonUnitDiag = true;
  bool verbose     = false;

  // User-provided parameters
  if(argc > 1) {
    m = atoi(argv[1]);
    n = atoi(argv[2]);
  }
  if(argc > 3) {
    sideLeft    = (bool) atoi(argv[3]);
    lower       = (bool) atoi(argv[4]);
    noTranspose = (bool) atoi(argv[5]);
    nonUnitDiag = (bool) atoi(argv[6]);
  }
  if(argc > 7)
    verbose = (bool) atoi(argv[7]);

  // Initialization
  timeval timeStart, timeEnd;

  // Report parameters
  printf("========================================\n");
  printf("  SHIFTED TRIANGULAR SOLVE VALIDATION\n");
  printf("========================================\n");
  printf("m = %d\n", m);
  printf("n = %d\n", n);
  printf("\n");
  printf("BLAS Options\n");
  printf("----------------------------------------\n");  
  printf("sideLeft    = %d\n", sideLeft);
  printf("lower       = %d\n", lower);
  printf("noTranspose = %d\n", noTranspose);
  printf("nonUnitDiag = %d\n", nonUnitDiag);

  // Initialize memory on host
  datafloat *A = (datafloat*) malloc(m*m*sizeof(datafloat));
  datafloat *B = (datafloat*) malloc(m*n*sizeof(datafloat));
  datafloat *shifts = (datafloat*) malloc(n*sizeof(datafloat));
  datafloat *X = (datafloat*) malloc(m*n*sizeof(datafloat));
  datafloat *residual = (datafloat*) malloc(m*n*sizeof(datafloat));

  // Initialize matrices on host
  datafloat alpha = randn<datafloat>();
  choleskyRandomMatrix<datafloat>(m,A);
#pragma omp parallel for
  for(int i=0;i<m*n;++i)
    B[i] = randn<datafloat>();
#pragma omp parallel for
  for(int i=0;i<n;++i) {
    shifts[i] = randn<datafloat>();
  }

  // Initialize memory on device
  datafloat *cuda_A, *cuda_B, *cuda_B_cublas, *cuda_shifts;
  hipMalloc(&cuda_A, m*m*sizeof(datafloat));
  hipMalloc(&cuda_B, m*n*sizeof(datafloat));
  hipMalloc(&cuda_B_cublas, m*n*sizeof(datafloat));
  hipMalloc(&cuda_shifts, n*sizeof(datafloat));
  hipMemcpy(cuda_A, A, m*m*sizeof(datafloat), hipMemcpyHostToDevice);
  hipMemcpy(cuda_B, B, m*n*sizeof(datafloat), hipMemcpyHostToDevice);
  hipMemcpy(cuda_B_cublas, B, m*n*sizeof(datafloat), hipMemcpyHostToDevice);
  hipMemcpy(cuda_shifts, shifts, n*sizeof(datafloat), hipMemcpyHostToDevice);

  // Initialize cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Initialize BLAS options
  hipblasSideMode_t  side  = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t  uplo  = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = HIPBLAS_OP_N;
  hipblasDiagType_t  diag  = HIPBLAS_DIAG_NON_UNIT;
  if(!sideLeft) {
    side = HIPBLAS_SIDE_RIGHT;
  }
  if(!lower) {
    uplo = HIPBLAS_FILL_MODE_UPPER;
  }
  if(!noTranspose) {
    trans = HIPBLAS_OP_T;
  }
  if(!nonUnitDiag) {
    diag = HIPBLAS_DIAG_UNIT;
  }

  // -------------------------------------------------
  // Test cudaMultiShiftTrsm
  // -------------------------------------------------

  // Solve triangular system
  hipDeviceSynchronize();
  gettimeofday(&timeStart, NULL);
  cudaMstrsm::cudaMultiShiftTrsm<datafloat>(handle, side, uplo, trans, diag, m, n,
					    &alpha, cuda_A, m, cuda_B, m, cuda_shifts);
  hipDeviceSynchronize();
  gettimeofday(&timeEnd, NULL);
  double cudaMstrsmTime
    = timeEnd.tv_sec - timeStart.tv_sec
    + (timeEnd.tv_usec - timeStart.tv_usec)/1e6;

  // Transfer result to host
  hipMemcpy(X, cuda_B, m*n*sizeof(datafloat), 
	     hipMemcpyDeviceToHost);

  // -------------------------------------------------
  // cuBLAS triangular solve
  //   For performance comparison
  // -------------------------------------------------

  // Solve triangular system
  hipDeviceSynchronize();
  gettimeofday(&timeStart, NULL);
  cublasTrsm<datafloat>(handle,side,uplo,trans,diag,m,n,
			&alpha,cuda_A,m,cuda_B_cublas,m);
  hipDeviceSynchronize();
  gettimeofday(&timeEnd, NULL);
  double cublasTime
    = timeEnd.tv_sec - timeStart.tv_sec
    + (timeEnd.tv_usec - timeStart.tv_usec)/1e6;

  // -------------------------------------------------
  // Output results
  // -------------------------------------------------

  // Report time for matrix multiplication
  printf("\n");
  printf("Timings\n");
  printf("----------------------------------------\n");
  printf("  cudaMstrsm : %g sec\n",cudaMstrsmTime);
  printf("  cuBLAS     : %g sec\n",cublasTime);

  // Report FLOPS
  double gflopCount = 1e-9*m*m*n; // Approximate
  printf("\n");
  printf("Performance\n");
  printf("----------------------------------------\n");
  printf("  cudaMstrsm : %g GFLOPS\n", gflopCount/cudaMstrsmTime);
  printf("  cuBLAS     : %g GFLOPS\n", gflopCount/cublasTime);
  
  if(verbose) {
    // Print matrices
    printf("\n");
    printf("Matrix entries\n");
    printf("----------------------------------------\n");
    printf("  alpha = %g\n", alpha);
    printf("  shifts = [");
    for(int i=0;i<n;++i)
      printf("%g ", shifts[i]);
    printf("]\n");
    printf("  A =\n    [[");
    for(int i=0;i<m;++i) {
      for(int j=0;j<=i;++j)
	printf("%g ", A[IDX(i,j,m)]);
      for(int j=i+1;j<m;++j)
	printf("0 ");
      printf("]");
      if(i<m-1)
	printf("\n    [");
      else
	printf("]\n");
    }
    printf("  B =\n    [[");
    for(int i=0;i<m;++i) {
      for(int j=0;j<n;++j)
	printf("%g ", B[IDX(i,j,m)]);
      printf(" ]");
      if(i<m-1)
	printf("\n    [");
      else
	printf("]\n");
    }
    printf("  X =\n    [[");
    for(int i=0;i<m;++i) {
      for(int j=0;j<n;++j)
	printf("%g ", X[IDX(i,j,m)]);
      printf(" ]");
      if(i<m-1)
	printf("\n    [");
      else
	printf("]\n");
    }
  }

  // Check error in solution
  double normB = nrm2<datafloat>(m*n,B,1);
  memcpy(residual,X,m*n*sizeof(datafloat));
  trmm<datafloat>('L','L','N','N',m,n,1,A,m,residual,m);
#pragma omp parallel for
  for(int i=0;i<n;++i)
    axpy<datafloat>(m, shifts[i], X+i*m, 1, residual+i*m, 1);
  axpy<datafloat>(m*n, -alpha, B, 1, residual, 1);
  double relResidual = nrm2<datafloat>(m*n,residual,1)/normB;
  printf("\n");
  printf("Relative error (Frobenius norm)\n");
  printf("----------------------------------------\n");
  printf("  cudaMstrsm : %g\n", relResidual);

  // -------------------------------------------------
  // Clean up and finish
  // -------------------------------------------------
  free(A);
  free(B);
  free(shifts);
  free(X);
  free(residual);
  hipFree(cuda_A);
  hipFree(cuda_B);
  hipFree(cuda_B_cublas);
  hipFree(cuda_shifts);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;

}
